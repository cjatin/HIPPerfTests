#include "hip/hip_runtime.h"
#define ENABLEGPU
#include <benchmark/benchmark.h>

__global__ void add(int* a) { *a += 3; }

static void BM_CudaMalloc(benchmark::State& state) {
  for (auto _ : state) {
    BENCHMARK_GPU_INIT();
    BENCHMARK_GPU_START();
    int *d_a;
    hipMalloc(&d_a, sizeof(int));
    hipFree(d_a);
    BENCHMARK_GPU_STOP();
  }
}
// Register the function as a benchmark
GPUBENCHMARK(BM_CudaMalloc);

// Define another benchmark
static void BM_LaunchKernel(benchmark::State& state) {
  for (auto _ : state) {
    BENCHMARK_GPU_INIT();
    BENCHMARK_GPU_START();
    int *d_a;
    hipMalloc(&d_a, sizeof(int));
    add<<<1,1>>>(d_a);
    hipFree(d_a);
    BENCHMARK_GPU_STOP();
  }
}
GPUBENCHMARK(BM_LaunchKernel);

BENCHMARK_MAIN();
